#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <windows.h>

typedef struct {
  int width;
  int height;
  int stride;
  float* elements;
} Matrix;

#define WIDTH_A 1024	
#define HEIGHT_A 1024

#define WIDTH_B 1024
#define HEIGHT_B 1024

#define BLOCK_SIZE 16
#define EPSILON 0.000001

void randMatInit(Matrix A)
{
	int row, column;
	
	for( row = 0; row < A.width; ++row)
	{
		for( column = 0; column < A.height; ++column)
		{
			A.elements[row * A.width + column] = ((float)(rand() % 1000)/10.0);
		}
	}
}

__device__ float GetElement(const Matrix A, int row, int col)
{
	return A.elements[A.stride * row + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value)
{
	A.elements[A.stride * row + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
	Matrix Asub;
	Asub.width = BLOCK_SIZE;
	Asub.height = BLOCK_SIZE;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE *col];
	
	return Asub;
}

__global__ void MatMulKernel_Share(const Matrix A, const Matrix B, Matrix C)
{
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x; 
	
	Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
	
	float pValue = 0.0;
	
	int row = threadIdx.y;
	int col = threadIdx.x;
	
	for(int index = 0; index < (A.width / BLOCK_SIZE); ++index)
	{	
		Matrix Asub = GetSubMatrix(A, blockRow, index);
		Matrix Bsub = GetSubMatrix(B, index, blockCol);
		
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
		
		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);
		
		__syncthreads();// Synchronize to make sure the sub-matrices As and Bs are loaded over before starting the computation
		
		for(int subIndex = 0 ; subIndex < BLOCK_SIZE; ++subIndex)
		{
			pValue += As[row][subIndex] * Bs[subIndex][col];
		}
		
		__syncthreads(); // Synchronize to make sure the computation completed for all threads before next load and computation round
	}
	
	SetElement(Csub, row, col, pValue);
}

int MatCompare(const Matrix A, const Matrix B)
{
	int i, j, index;
	if(A.height != B.height || A.width != B.width)
		return 0;
	for(i = 0; i < A.height; i++)
		for(j = 0; j < A.width; j++) {
			index = i * A.width + j;
			if(fabsf(A.elements[index] - B.elements[index]) > EPSILON)
				return 0;
		}
	return 1;
}

void MatMul_CPU(const Matrix A, const Matrix B, Matrix C)
{
	int i, j, k;
	for(i = 0; i < C.width; i++)
		for(j = 0; j < C.height; j++) {
			float sum = 0;
			for(k = 0; k < A.width; k++)
				sum += A.elements[i * A.width + k] * B.elements[k * B.width + j];
			C.elements[i * C.width + j] = sum;
		}
}


void MatMul_GPU(const Matrix A, const Matrix B, Matrix C)
{
	Matrix d_A, d_B, d_C;
	
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	d_A.width = A.width;
	d_A.height = A.height;
	d_A.stride = d_A.width;

	d_B.width = B.width;
	d_B.height = B.height;
	d_B.stride = d_B.width;
	
	d_C.width = B.width;
	d_C.height = A.height;
	d_C.stride = d_B.width;
	
	hipMalloc(&(d_A.elements), d_A.width * d_A.height * sizeof(float));
	hipMemcpy(d_A.elements, A.elements, d_A.width * d_A.height * sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc(&(d_B.elements), d_B.width *d_B.height * sizeof(float));
	hipMemcpy(d_B.elements, B.elements, d_B.width *d_B.height * sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc(&(d_C.elements), d_C.width * d_C.height * sizeof(float));

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // each block has BLOCK_SIZE * BLOCK_SIZE threads with two dimensions.
	
	dim3 dimGrid(ceil(1.0 * d_C.height / dimBlock.x), ceil(1.0 * d_C.width / dimBlock.y));
	
	hipEventRecord(start);
	MatMulKernel_Share<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&elapsedTime, start, stop);
		
	hipMemcpy(C.elements, d_C.elements, d_C.width * d_C.height * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	printf("Matrix Multiplication time on GPU is: %fms\n", elapsedTime);
}

int main()
{
	hipError_t cudaStatus;
	Matrix A, B, C, D;
	DWORD start, end;

	A.width = WIDTH_A;
	A.height = HEIGHT_A;
	
	B.width = WIDTH_B;
	B.height = HEIGHT_B;
	
	C.width = B.width;
	C.height = A.height;

	D.width = B.width;
	D.height = A.height;

	A.elements = (float *)malloc(sizeof(float) * A.width * A.height);
	B.elements = (float *)malloc(sizeof(float) * B.width * B.height);
	C.elements = (float *)malloc(sizeof(float) * C.width * C.height);
	D.elements = (float *)malloc(sizeof(float) * D.width * D.height);
	
	randMatInit(A);
	randMatInit(B);

	/* Run matrix multiplication on CPU: C = A x B */
	start = GetTickCount();
	MatMul_CPU(A, B, C);
	end = GetTickCount();
	printf("Matrix Multiplication time on GPU is: %dms\n", end-start);

	/* Run matrix multiplication on GPU: D = A x B */
	MatMul_GPU(A, B, D);

	/* verify the results */
	if(!MatCompare(C, D))
		printf("Your GPU results are different from CPU results.\n");
	else
		printf("Your GPU results are the same as CPU results.\n");

	free(A.elements);
	free(B.elements);
	free(C.elements);
	free(D.elements);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
