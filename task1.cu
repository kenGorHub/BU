#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <windows.h>

typedef struct {
  int width;
  int height;
  float* elements;
} Matrix;

#define WIDTH_A 2048
#define HEIGHT_A 2048

#define WIDTH_B 2048
#define HEIGHT_B 2048

#define BLOCK_SIZE 16
#define EPSILON 0.000001

void randMatInit(Matrix A)
{
	int row, column;
	
	for( row = 0; row < A.width; ++row)
	{
		for( column = 0; column < A.height; ++column)
		{
			A.elements[row * A.width + column] = ((float)(rand() % 1000)/10.0);
		}
	}
}


__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C)
{
	int column = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	
	float pValue = 0.0;
	
	for(int index = 0; index < A.width; ++index)
	{
		pValue = A.elements[row * A.width + index] * B.elements[index * B.width + column];
	}
	
	C.elements[row * C.width + column] = pValue;
}

int MatCompare(const Matrix A, const Matrix B)
{
	int i, j, index;
	if(A.height != B.height || A.width != B.width)
		return 0;
	for(i = 0; i < A.height; i++)
		for(j = 0; j < A.width; j++) {
			index = i * A.width + j;
			if(fabsf(A.elements[index] - B.elements[index]) > EPSILON)
				return 0;
		}
	return 1;
}

void MatMul_CPU(const Matrix A, const Matrix B, Matrix C)
{
	int i, j, k;
	for(i = 0; i < C.width; i++)
		for(j = 0; j < C.height; j++) {
			float sum = 0;
			for(k = 0; k < A.width; k++)
				sum += A.elements[i * A.width + k] * B.elements[k * B.width + j];
			C.elements[i * C.width + j] = sum;
		}
}


void MatMul_GPU(const Matrix A, const Matrix B, Matrix C)
{
	Matrix d_A, d_B, d_C;
	
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	d_A.width = A.width;
	d_A.height = A.height;
	
	d_B.width = B.width;
	d_B.height = B.height;
	
	d_C.width = B.width;
	d_C.height = A.height;
	
	hipMalloc(&(d_A.elements), d_A.width * d_A.height * sizeof(float));
	hipMemcpy(d_A.elements, A.elements, d_A.width * d_A.height * sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc(&(d_B.elements), d_B.width *d_B.height * sizeof(float));
	hipMemcpy(d_B.elements, B.elements, d_B.width *d_B.height * sizeof(float), hipMemcpyHostToDevice);
	
	hipMalloc(&(d_C.elements), d_C.width * d_C.height * sizeof(float));

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // each block has BLOCK_SIZE * BLOCK_SIZE threads with two dimensions.
	
	dim3 dimGrid(ceil(1.0 * d_C.height / dimBlock.x), ceil(1.0 * d_C.width / dimBlock.y));
	
	hipEventRecord(start);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&elapsedTime, start, stop);
		
	hipMemcpy(C.elements, d_C.elements, d_C.width * d_C.height * sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	printf("Matrix Multiplication time on GPU is: %fms\n", elapsedTime);
}

int main()
{
	hipError_t cudaStatus;
	Matrix A, B, C, D;
	DWORD start, end;

	A.width = WIDTH_A;
	A.height = HEIGHT_A;
	
	B.width = WIDTH_B;
	B.height = HEIGHT_B;
	
	C.width = B.width;
	C.height = A.height;

	D.width = B.width;
	D.height = A.height;

	A.elements = (float *)malloc(sizeof(float) * A.width * A.height);
	B.elements = (float *)malloc(sizeof(float) * B.width * B.height);
	C.elements = (float *)malloc(sizeof(float) * C.width * C.height);
	D.elements = (float *)malloc(sizeof(float) * D.width * D.height);
	
	randMatInit(A);
	randMatInit(B);

	/* Run matrix multiplication on CPU: C = A x B */
	start = GetTickCount();
	MatMul_CPU(A, B, C);
	end = GetTickCount();
	printf("Matrix Multiplication time on GPU is: %dms\n", end-start);

	/* Run matrix multiplication on GPU: D = A x B */
	MatMul_GPU(A, B, D);

	/* verify the results */
	if(!MatCompare(C, D))
		printf("Your GPU results are different from CPU results.\n");
	else
		printf("Your GPU results are the same as CPU results.\n");

	free(A.elements);
	free(B.elements);
	free(C.elements);
	free(D.elements);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
